#include "hip/hip_runtime.h"
// This file is part of GElib, a C++/CUDA library for group
// equivariant tensor operations. 
// 
// Copyright (c) 2022s, Imre Risi Kondor
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.


#ifndef _SO3Fpart_addFproduct_cu
#define _SO3Fpart_addFproduct_cu

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <thrust/complex.h>
//#include <thrust/tuple.h>

#include "SO3_CGbank.hpp"
#include "Ctensor2_view.hpp"
#include "Ctensor3_view.hpp"

//__device__ __constant__ unsigned char cg_cmem[32276]; 

extern GElib::SO3_CGbank SO3_cgbank;




__device__ int loadg3(const cnine::Ctensor3_view& x, float* dest, const int b, const int t){
  int I=x.n1;
  int J=x.n2;
  int s1=x.s1;
  int s2=x.s2;
  int offs=I*J; //((I*J-1)/32+1)*32;
  float* destc=dest+offs;
  float* source=x.arr+x.s0*b;
  float* sourcec=x.arrc+x.s0*b;
  if(t<J){
    for(int i=0; i<I; i++)
      dest[i*J+t]=source[i*s1+t*s2];
    for(int i=0; i<I; i++)
      destc[i*J+t]=sourcec[i*s1+t*s2];
  }
  return offs;
}


__device__ int saveg3(const cnine::Ctensor3_view& x, float* source, const int b, const int t){
  int I=x.n1;
  int J=x.n2;
  int s1=x.s1;
  int s2=x.s2;
  int offs=I*J; //((I*J-1)/32+1)*32;
  float* sourcec=source+offs;
  float* dest=x.arr+x.s0*b;
  float* destc=x.arrc+x.s0*b;
  if(t<J){
    for(int i=0; i<I; i++)
      dest[i*s1+t*s2]=source[i*J+t];
    for(int i=0; i<I; i++)
      destc[i*s1+t*s2]=sourcec[i*J+t];
  }
  return offs;
}

__device__ int loadg3c(const cnine::Ctensor3_view& x, float* dest, const int b, const int t){
  int I=x.n1;
  int J=x.n2;
  int s1=x.s1;
  int s2=x.s2;
  int offs=I*J; //((I*J-1)/32+1)*32;
  float* destc=dest+offs;
  float* source=x.arr+x.s0*b;
  float* sourcec=x.arrc+x.s0*b;
  if(t<J){
    for(int i=0; i<I; i++)
      dest[i*J+t]=source[i*s1+t*s2];
    for(int i=0; i<I; i++)
      destc[i*J+t]=-sourcec[i*s1+t*s2];
  }
  return offs;
}


/*
__device__ int saveg3c(const cnine::Ctensor3_view& x, float* source, const int b, const int t){
  int I=x.n1;
  int J=x.n2;
  int s1=x.s1;
  int s2=x.s2;
  int offs=I*J; //((I*J-1)/32+1)*32;
  float* sourcec=source+offs;
  float* dest=x.arr+x.s0*b;
  float* destc=x.arrc+x.s0*b;
  if(t<J){
    for(int i=0; i<I; i++)
      dest[i*s1+t*s2]=source[i*J+t];
    for(int i=0; i<I; i++)
      destc[i*s1+t*s2]=-sourcec[i*J+t];
  }
  return offs;
}
*/


__global__ void SO3Fpart_addFproduct_kernel(const cnine::Ctensor3_view r, const cnine::Ctensor3_view x, 
  const cnine::Ctensor3_view y, const int Cptr, float* cptr_global, const int conj){

  extern __shared__ unsigned char _shared[]; 
  //const float* C_ptr=reinterpret_cast<float*>(cg_cmem)+Cptr;
  const int b=blockIdx.x;
  const int t=threadIdx.x;

  int l1=(x.n1-1)/2;
  int l2=(y.n1-1)/2;
  int l=(r.n1-1)/2;
  int xn=x.n2;
  int yn=y.n2;
  int rn=r.n2;

  float* cptr;
  if(Cptr>=0) cptr=reinterpret_cast<float*>(cg_cmem)+Cptr;
  else cptr=cptr_global;

  float* xpr=reinterpret_cast<float*>(_shared);
  float* xpi=xpr+x.n1*x.n2;
  loadg3(x,xpr,b,t);

  float* ypr=xpr+((2*xn*xn-1)/32+1)*32;
  float* ypi=ypr+y.n1*y.n2;
  if(conj==0) loadg3(y,ypr,b,t);
  else loadg3c(y,ypr,b,t);

  float* rpr=ypr+((2*yn*yn-1)/32+1)*32;
  float* rpi=rpr+r.n1*r.n2;
  loadg3(r,rpr,b,t);

  __syncthreads();

  if(t<xn*yn){

    int i1=t/yn;
    xpr=xpr+i1;
    xpi=xpi+i1;
    
    int i2=t%yn;
    ypr=ypr+i2;
    ypi=ypi+i2;
    
    int i=i1+i2-l1-l2+l;
    float* _rpr=rpr+i;
    float* _rpi=rpi+i;

    if(i>=0 && i<rn){

      float c0=cptr[i1*yn+i2]*xn*yn/rn;
      
      for(int m1=-l1; m1<=l1; m1++){
	const float x_r=xpr[xn*(m1+l1)];
	const float x_i=xpi[xn*(m1+l1)];
	int lower=-l-m1; if(lower<-l2) lower=-l2;
	int upper=l-m1; if(upper>l2) upper=l2;
	for(int m2=lower; m2<=upper; m2++){
	  float c=cptr[(m1+l1)*yn+m2+l2];
	  const float y_r=ypr[yn*(m2+l2)];
	  const float y_i=ypi[yn*(m2+l2)];
	  //_rpr[rn*(m1+m2+l)]+=c0*c*(x_r*y_r-x_i*y_i); 
	  //_rpi[rn*(m1+m2+l)]+=c0*c*(x_r*y_i+x_i*y_r);
	  atomicAdd(_rpr+rn*(m1+m2+l),c0*c*(x_r*y_r-x_i*y_i)); 
	  atomicAdd(_rpi+rn*(m1+m2+l),c0*c*(x_r*y_i+x_i*y_r));
	}
 
      }
    }
  }

  __syncthreads();
  
  saveg3(r,rpr,b,t);

}



namespace GElib{


  void SO3Fpart_addFproduct_cu(const cnine::Ctensor3_view& r, const cnine::Ctensor3_view& x, const cnine::Ctensor3_view& y, 
    const int conj,const hipStream_t& stream){

    const int xl=(x.n1-1)/2;
    const int yl=(y.n1-1)/2;
    const int l=(r.n1-1)/2;
    const int b=r.n0;

    float* cptr=nullptr;
    int Cptr=SO3_cgbank.getfC(xl,yl,l)/4;
    if(Cptr<0) cptr=SO3_cgbank.getf(CGindex(xl,yl,l),r.dev).arrg;
    int clines=cnine::roundup(x.n1*y.n1,32)/32;

    int nlines=cnine::roundup(x.n1*x.n2*2,32)/32+
      cnine::roundup(y.n1*y.n2*2,32)/32+
      cnine::roundup(r.n1*r.n2*2,32)/32;

    if(nlines<=384){
      SO3Fpart_addFproduct_kernel<<<b,cnine::roundup(x.n2*y.n2,32),nlines*128,stream>>>
	(r,x,y,Cptr,cptr,conj);
      return; 
    }

    cout<<"error"<<endl;

  }    


}


#endif 

